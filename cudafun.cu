#ifndef DUDAFUN
#define DUDAFUN

#include <iostream>


void cuda_info()
{
    int dev = 0;
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, dev);
    std::cout << "使用GPU device " << dev << ": " << devProp.name << std::endl;
    std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
    std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
    std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
    std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "每个SM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;

}

static __global__ void add(float* x, float * y, float* z, int n)
{
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
    {
        z[i] = x[i] + y[i];
    }
}

// 不要在cpp文件中include以 .cu 文件，因为遇到<<< >>> 这样的符号容易编译不过
// 一般将所有的 .cu 生成动态库，然后再连接该库

void add_run(float* x, float * y, float* z, int n)
{
    dim3 blockSize(256);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
    add<<<gridSize, blockSize>>>(x, y, z, n);
}

#endif